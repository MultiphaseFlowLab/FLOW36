#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <complex.h>

#include "cuda_variables.h"
#include "free_gpu.h"

extern "C" void h_free_gpu(int spx, int spy, int nz, int nx)
//-------------------------------------------------------------------------------------
//
//     Reset the GPU and free allocated memory
//
//     Copyright Multiphase Flow Laboratory, University of Udine
//     authors - D. Di Giusto, Jan 2020
//
//-------------------------------------------------------------------------------------
{

#define phiflag phicompflag
#define psiflag psicompflag
#define expx expansionx
#define expy expansiony
#define expz expansionz


  ok = 0;
  ok = ok + hipFree(d_batch);
  ok = ok + hipFree(d_batch_c);

  ok = ok + hipFree(ur_d);
  ok = ok + hipFree(uc_d);

  ok = ok + hipFree(d_uopr);
  ok = ok + hipFree(d_uopc);

  if (ok!=0) printf("Failure in freeing the arrays!!\n");

#if phiflag == 1 || psiflag == 1 || expx != 1 || expy != 1 || expz != 1
  //fg
  ok = ok + hipFree(d_phic);
  ok = ok + hipFree(d_phic_c);
  ok = ok + hipFree(psir_d);
  ok = ok + hipFree(psic_d);
  ok = ok + hipFree(d_psir);
  ok = ok + hipFree(d_psic);
  if (ok!=0) printf("Failure in freeing the fg arrays!!\n");

  ok = ok + hipfftDestroy(plan_x_psi);
  ok = ok + hipfftDestroy(plan_z_psi);
  ok = ok + hipfftDestroy(plan_x_fwd_psi);
  ok = ok + hipfftDestroy(plan_y_many_psi);
  if (ok!=0) printf("Failure in fg plans destruction!!\n");

#endif

  ok = 0;
  ok = ok + hipfftDestroy(plan_x);
  ok = ok + hipfftDestroy(plan_y);
  ok = ok + hipfftDestroy(plan_z);
  ok = ok + hipfftDestroy(plan_x_fwd);
  ok = ok + hipfftDestroy(plan_y_many);

  if (ok!=0) printf("Error in planes destruction!!\n");

  hipDeviceReset();
}//end subroutine h_free_gpu
//*************************************************************************************
//
//
//
//
//
//
//
//
//
//
//*************************************************************************************
