#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <complex.h>

#include "cuda_variables.h"
#include "free_gpu.h"

extern "C" void h_free_gpu(int spx, int spy, int nz, int nx)
//-------------------------------------------------------------------------------------
//
//     Reset the GPU and free allocated memory
//
//     Copyright Multiphase Flow Laboratory, University of Udine
//     authors - D. Di Giusto, Jan 2020
//
//-------------------------------------------------------------------------------------
{

  ok = 0;
  ok = ok + hipFree(d_batch);
  ok = ok + hipFree(d_batch_c);

  ok = ok + hipFree(ur_d);
  ok = ok + hipFree(uc_d);

  ok = ok + hipFree(d_uopr);
  ok = ok + hipFree(d_uopc);

  if (ok!=0) printf("Failure in freeing the arrays!!\n");

  ok = ok + hipfftDestroy(plan_x);
  ok = ok + hipfftDestroy(plan_y);
  ok = ok + hipfftDestroy(plan_z);
  ok = ok + hipfftDestroy(plan_x_fwd);
  ok = ok + hipfftDestroy(plan_y_many);

  if (ok!=0) printf("Error in planes destruction!!\n");

  hipDeviceReset();
}//end subroutine h_free_gpu
//*************************************************************************************
//
//
//
//
//
//
//
//
//
//
//*************************************************************************************
