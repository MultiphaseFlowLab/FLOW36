#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <complex.h>

#include "cuda_variables.h"
#include "free_gpu.h"

extern "C" void h_free_gpu(int spx, int spy, int nz, int nx)
//-------------------------------------------------------------------------------------
//
//     Reset the GPU and free allocated memory
//
//     Copyright Multiphase Flow Laboratory, University of Udine
//     authors - D. Di Giusto, Jan 2020
//
//-------------------------------------------------------------------------------------
{

  ok = 0;
  ok = ok + hipFree(d_batch);
  ok = ok + hipFree(d_batch_c);

  ok = ok + hipFree(ur_d);
  ok = ok + hipFree(uc_d);
//  ok = ok + hipFree(d_uout);

  ok = ok + hipFree(d_uopr);
  ok = ok + hipFree(d_uopc);
//  ok = ok + hipFree(d_uext);

  //  ok = ok + hipFree(vr_d);
//  ok = ok + hipFree(vc_d);
//  ok = ok + hipFree(d_vout);

//  ok = ok + hipFree(wr_d);
//  ok = ok + hipFree(wc_d);
//  ok = ok + hipFree(d_wout);

//  ok = ok + hipFree(d_uu);

//  ok = ok + hipFree(dz_d);
//  ok = ok + hipFree(fstart_d);
  if (ok!=0) printf("Failure in freeing the arrays!!\n");

  ok = ok + hipfftDestroy(plan_x);
  ok = ok + hipfftDestroy(plan_y);
  ok = ok + hipfftDestroy(plan_z);
  ok = ok + hipfftDestroy(plan_x_fwd);
//  ok = ok + hipfftDestroy(plan_y_fwd);
  ok = ok + hipfftDestroy(plan_y_many);
//  ok = ok + hipfftDestroy(plan_z_fwd);

  if (ok!=0) printf("Error in planes destruction!!\n");

  hipDeviceReset();
}//end subroutine h_free_gpu
//*************************************************************************************
//
//
//
//
//
//
//
//
//
//
//*************************************************************************************
